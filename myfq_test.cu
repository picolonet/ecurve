#include "hip/hip_runtime.h"
#include <algorithm>
#include <cstdio>
#include <cstdint>
#include <cstring>
#include <cassert>
#include <vector>

#include <time.h>

#include <gmp.h>

#include <libff/algebra/curves/mnt753/mnt4753/mnt4753_pp.hpp>
#include <libff/algebra/curves/mnt753/mnt6753/mnt6753_pp.hpp>

#include "constants.h"

#include "myfq.cu"
#include "utils.cu"

const char* input_a = "/home/arunesh/github/snark-challenge/reference-01-field-arithmetic/inputs";

using namespace libff;

void test_fq_add(std::vector<uint8_t*> x, std::vector<uint8_t*> y, int num_bytes, FILE* debug_file);
void test_fq_sub(std::vector<uint8_t*> x, std::vector<uint8_t*> y, int num_bytes, FILE* debug_file);
void test_fq_mul_byconst(std::vector<uint8_t*> x, std::vector<uint8_t*> y, int num_bytes, FILE* debug_file);
void test_fq_mont_mul(std::vector<uint8_t*> x, std::vector<uint8_t*> y, int num_bytes, FILE* debug_file);
void test_fq_sub_newtest(std::vector<uint8_t*> x, std::vector<uint8_t*> y, int num_bytes, FILE* debug_file);

struct fq_op { // Helper function to ease cleanup of container
    void operator () (std::vector<uint8_t*> x, std::vector<uint8_t*> y, int num_bytes, FILE* debug_file) ;
};

struct add_fq_op : fq_op {
    void operator () (std::vector<uint8_t*> x, std::vector<uint8_t*> y, int num_bytes, FILE* debug_file) {
        test_fq_add(x, y, num_bytes, debug_file);
    } 
};

struct sub_fq_op : fq_op {
    void operator () (std::vector<uint8_t*> x, std::vector<uint8_t*> y, int num_bytes, FILE* debug_file) {
        test_fq_sub(x, y, num_bytes, debug_file);
    } 
};

struct sub_newtest: fq_op {
    void operator () (std::vector<uint8_t*> x, std::vector<uint8_t*> y, int num_bytes, FILE* debug_file) {
        test_fq_sub_newtest(x, y, num_bytes, debug_file);
    } 
};

struct mul_by13_fq_op : fq_op {
    void operator () (std::vector<uint8_t*> x, std::vector<uint8_t*> y, int num_bytes, FILE* debug_file) {
        test_fq_mul_byconst(x, y, num_bytes, debug_file);
    } 
};

struct mont_mul_fq_op : fq_op {
    void operator () (std::vector<uint8_t*> x, std::vector<uint8_t*> y, int num_bytes, FILE* debug_file) {
        test_fq_mont_mul(x, y, num_bytes, debug_file);
    } 
};

uint8_t* read_mnt_fq_2_gpu(FILE* inputs) {
  uint8_t* buf; 
  hipMallocManaged(&buf, bytes_per_elem , sizeof(uint8_t));
  // the input is montgomery representation x * 2^768 whereas cuda-fixnum expects x * 2^1024 so we shift over by (1024-768)/8 bytes
  fread((void*)buf, io_bytes_per_elem*sizeof(uint8_t), 1, inputs);
  return buf;
}

template <typename fqop_temp>
void loadrun_fq_op(const char* input_file, const char* debug_filename) {
  // argv[2] = input_a;
  auto inputs = fopen(input_file, "r");
  auto debug_file = fopen(debug_filename, "w");
  printf("\n Opening file %s for reading.\n", input_a);

  size_t n;
  clock_t start, end;
  double time_used = 0.0;
  double time_iter = 0.0;

  fprintf(debug_file, "\n mnt4 modulus:\n");
  fprint_uint8_array(debug_file, mnt4_modulus, bytes_per_elem); 

  printf("\n Size of mplimb_t = %d, %d, %d", sizeof(mp_limb_t), sizeof(mp_size_t), libff::mnt4753_q_limbs);

  while(true) {
  
    size_t array_size = fread((void*) &n, sizeof(size_t), 1, inputs);
    if (array_size == 0) break;
    printf("\n Array size = %d\n", n);
    std::vector<uint8_t*> x;
    std::vector<uint8_t*> y;
    for (size_t i = 0; i < n; ++i) {
      uint8_t* ptr = read_mnt_fq_2(inputs);
      x.emplace_back(ptr);
    }
    for (size_t i = 0; i < n; ++i) {
      y.emplace_back(read_mnt_fq_2(inputs));
    }
    std::vector<uint8_t*> x6;
    std::vector<uint8_t*> y6;
    for (size_t i = 0; i < n; ++i) {
      x6.emplace_back(read_mnt_fq_2(inputs));
    }
    for (size_t i = 0; i < n; ++i) {
      y6.emplace_back(read_mnt_fq_2(inputs));
    }

    start = clock();
    fqop_temp()(x, y, bytes_per_elem, debug_file);
    end = clock();

    time_iter = ((double) end-start) * 1000.0 / CLOCKS_PER_SEC;
    time_used += time_iter;
    printf("\n Full test function Round N, time = %5.4f ms.\n", time_iter); 
 
    std::for_each(x.begin(), x.end(), delete_ptr());
    x.clear();
    std::for_each(y.begin(), y.end(), delete_ptr());
    y.clear();
    std::for_each(x6.begin(), x6.end(), delete_ptr());
    x6.clear();
    std::for_each(y6.begin(), y6.end(), delete_ptr());
    y6.clear();
  } 
  fclose(inputs);
  fclose(debug_file);
}


void loadrun_fq_add(const char* input_file, const char* debug_filename) {
  // argv[2] = input_a;
  auto inputs = fopen(input_file, "r");
  auto debug_file = fopen(debug_filename, "w");
  printf("\n Opening file %s for reading.\n", input_a);

  size_t n;
  clock_t start, end;
  double time_used = 0.0;
  double time_iter = 0.0;

  fprintf(debug_file, "\n mnt4 modulus:\n");
  fprint_uint8_array(debug_file, mnt4_modulus, bytes_per_elem); 

  printf("\n Size of mplimb_t = %d, %d, %d", sizeof(mp_limb_t), sizeof(mp_size_t), libff::mnt4753_q_limbs);

  while(true) {
  
    size_t array_size = fread((void*) &n, sizeof(size_t), 1, inputs);
    if (array_size == 0) break;
    printf("\n Array size = %d\n", n);
    std::vector<uint8_t*> x;
    std::vector<uint8_t*> y;
    std::vector<uint8_t*> z;
    for (size_t i = 0; i < n; ++i) {
      uint8_t* ptr = read_mnt_fq_2(inputs);
      uint8_t* ptr2 = (uint8_t*)calloc(io_bytes_per_elem, sizeof(uint8_t));
      std::memcpy(ptr2, ptr, io_bytes_per_elem*sizeof(uint8_t));
      x.emplace_back(ptr);
      z.emplace_back(ptr2);
    }
    for (size_t i = 0; i < n; ++i) {
      y.emplace_back(read_mnt_fq_2(inputs));
    }
    std::vector<uint8_t*> x6;
    std::vector<uint8_t*> y6;
    for (size_t i = 0; i < n; ++i) {
      x6.emplace_back(read_mnt_fq_2(inputs));
    }
    for (size_t i = 0; i < n; ++i) {
      y6.emplace_back(read_mnt_fq_2(inputs));
    }

    int num_threads = io_bytes_per_elem / 8;

    start = clock();
    std::vector<uint8_t*>* result;
    test_fq_add(x, y, bytes_per_elem, debug_file);
    end = clock();

    time_iter = ((double) end-start) * 1000.0 / CLOCKS_PER_SEC;
    time_used += time_iter;
    printf("\n Full test function Round N, time = %5.4f ms.\n", time_iter); 
 
    std::for_each(x.begin(), x.end(), delete_ptr());
    x.clear();
    std::for_each(y.begin(), y.end(), delete_ptr());
    y.clear();
    std::for_each(x6.begin(), x6.end(), delete_ptr());
    x6.clear();
    std::for_each(y6.begin(), y6.end(), delete_ptr());
    y6.clear();

  
  } 
  fclose(inputs);
  fclose(debug_file);
}

// no modulus MPN add that uses libgmp.
void mpn_mul_u64(uint8_t* a, uint64_t mul, int size_bytes) {
   mpn_mul_1((mp_limb_t *)a, (const mp_limb_t*) a, size_bytes / sizeof(mp_limb_t), mul);
}

// We test basic big int addition by a0 + a1 for a fq2 element.
void test_fq_mul_byconst(std::vector<uint8_t*> x, std::vector<uint8_t*> y, int num_bytes, FILE* debug_file) {
  mnt4753_pp::init_public_params();
  mnt6753_pp::init_public_params();

  std::vector<Fq<mnt4753_pp>> x0;
  std::vector<Fq<mnt4753_pp>> x1;
  cgbn_error_report_t *report;
  NEW_CUDA_CHECK(cgbn_error_report_alloc(&report));

  int tpb = TPB;
  // printf("\n Threads per block =%d", tpb);
  int IPB = TPB/TPI;

  int n = x.size();
  single_mfq_ti* gpuInstances;
  single_mfq_ti* localInstances;
  fprintf(debug_file, "\n size of fq2_t:%d", sizeof(single_mfq_ti));
  localInstances = (single_mfq_ti*) calloc(n, sizeof(single_mfq_ti));
  NEW_CUDA_CHECK(hipSetDevice(0));
  NEW_CUDA_CHECK(hipMalloc((void **)&gpuInstances, sizeof(single_mfq_ti)*n));
  load_mnt4_modulus();
  
  for (int i = 0; i < n; i++) {
      std::memcpy((void*)localInstances[i].x, (void*)x[i], num_bytes);
      localInstances[i].carry = 0;
      //std::memcpy((void*)localInstances[i].y, (void*)y[i], num_bytes);
  }
  
  NEW_CUDA_CHECK(hipMemcpy(gpuInstances, localInstances, sizeof(single_mfq_ti) * n, hipMemcpyHostToDevice));
  //for (int i = 0; i < n; i++) {
  //    NEW_CUDA_CHECK(hipMemcpy(gpuInstances[i].a0, x[i], num_bytes, hipMemcpyHostToDevice));
  //    NEW_CUDA_CHECK(hipMemcpy(gpuInstances[i].a1, y[i], num_bytes, hipMemcpyHostToDevice));
  //}

  uint32_t num_blocks = (n + IPB-1)/IPB;
  clock_t start, end;
  double time_iter = 0.0;

  const uint64_t multiplier = 0x0FFFFFFFFFFFFFFFull;
  // const uint64_t multiplier = 0x0FFFull;

  n = 1; // CHANGE CHANGE
  start = clock();
  fq_mul_const_kernel<<<num_blocks, TPB>>>(gpuInstances, n, mnt4_modulus_device, multiplier);
  NEW_CUDA_CHECK(hipDeviceSynchronize());
  end = clock();
  time_iter = ((double) end-start) * 1000.0 / CLOCKS_PER_SEC;
  fprintf(debug_file, "\n num_elements = %d, compute ony latency = %8.7f ms, per element = %8.7f microseconds.\n", n,
      time_iter, 1000.0*time_iter / (double)n); 
  printf("\n num_elements = %d, compute ony latency = %8.7f ms, per element = %8.7f microseconds.\n", n,
      time_iter, 1000.0*time_iter / (double)n); 
  NEW_CUDA_CHECK(hipMemcpy(localInstances, gpuInstances, sizeof(single_mfq_ti) * n, hipMemcpyDeviceToHost));
 
  Fq<mnt4753_pp> const13; 
  const13.set_ulong(multiplier);
  for (int i = 0; i < n; i++) {
    Fq<mnt4753_pp> out;
    x0.emplace_back(to_fq(x[i]));
    x1.emplace_back(to_fq(y[i]));
    out = x0[i] * const13;
    mpn_mul_u64(x[i], multiplier, num_bytes); 
    fprintf(debug_file, "\n REF MUL BY CONST:\n");
    fprint_uint8_array(debug_file, (uint8_t*)x[i], num_bytes); 
    fprintf(debug_file, "\n MY MUL BY CONST, carry = %d:\n", localInstances[i].carry);
    fprint_uint8_array(debug_file, (uint8_t*)localInstances[i].x, num_bytes); 
    if (check((uint8_t*) out.mont_repr.data, (uint8_t*)localInstances[i].x, io_bytes_per_elem)) {
      printf("\n TEST FAILED.");
      fprintf(debug_file, "\n TEST FAILED.");
    }
  }

  // free memory
  hipFree(gpuInstances);
  free(localInstances);
}

// no modulus MPN add that uses libgmp.
void mpn_add(uint8_t* a, uint8_t* b, int size_bytes) {
   mpn_add_n((mp_limb_t *)a, (const mp_limb_t*) a, (const mp_limb_t*) b,
      size_bytes / sizeof(mp_limb_t));
}

// no modulus MPN sub A-B that uses libgmp. returns carry (borrow).
int mpn_sub(uint8_t* a, uint8_t* b, int size_bytes) {
   mpn_sub_n((mp_limb_t *)a, (const mp_limb_t*) a, (const mp_limb_t*) b,
      size_bytes / sizeof(mp_limb_t));
}

// We test basic big int addition by a0 + a1 for a fq2 element.
void test_fq_sub_newtest(std::vector<uint8_t*> x, std::vector<uint8_t*> y, int num_bytes, FILE* debug_file) {

  mnt4753_pp::init_public_params();
  mnt6753_pp::init_public_params();

  std::vector<Fq<mnt4753_pp>> x0;
  std::vector<Fq<mnt4753_pp>> x1;
  cgbn_error_report_t *report;
  NEW_CUDA_CHECK(cgbn_error_report_alloc(&report));

  int tpb = TPB;
  // printf("\n Threads per block =%d", tpb);
  int IPB = TPB/TPI;

  int n = x.size();
  tuple_mfq_ti* gpuInstances;
  tuple_mfq_ti* localInstances;
  fprintf(debug_file, "\n size of fq2_t:%d", sizeof(tuple_mfq_ti));
  localInstances = (tuple_mfq_ti*) calloc(n, sizeof(tuple_mfq_ti));
  NEW_CUDA_CHECK(hipSetDevice(0));
  NEW_CUDA_CHECK(hipMalloc((void **)&gpuInstances, sizeof(tuple_mfq_ti)*n));
  load_mnt4_modulus();

  uint8_t* z = (uint8_t*) calloc(num_bytes, sizeof(uint8_t));
  for (int i = 0; i < n; i++) {
    mpn_add(y[i], x[i], num_bytes);
    std::memcpy((void*)localInstances[i].x, (void*)x[i], num_bytes);
    std::memcpy((void*)localInstances[i].y, (void*)y[i], num_bytes);
    localInstances[i].carry = 0;
  }

  NEW_CUDA_CHECK(hipMemcpy(gpuInstances, localInstances, sizeof(tuple_mfq_ti) * n, hipMemcpyHostToDevice));
  //for (int i = 0; i < n; i++) {
  //    NEW_CUDA_CHECK(hipMemcpy(gpuInstances[i].a0, x[i], num_bytes, hipMemcpyHostToDevice));
  //    NEW_CUDA_CHECK(hipMemcpy(gpuInstances[i].a1, y[i], num_bytes, hipMemcpyHostToDevice));
  //}

  uint32_t num_blocks = (n + IPB-1)/IPB;
  clock_t start, end;
  double time_iter = 0.0;

  start = clock();
  fq_sub_nomod_kernel<<<num_blocks, TPB>>>(gpuInstances, 1, mnt4_modulus_device);
  NEW_CUDA_CHECK(hipDeviceSynchronize());
  end = clock();
  time_iter = ((double) end-start) * 1000.0 / CLOCKS_PER_SEC;
  fprintf(debug_file, "\n num_elements = %d, compute ony latency = %8.7f ms, per element = %8.7f microseconds.\n", n,
      time_iter, 1000.0*time_iter / (double)n); 
  printf("\n num_elements = %d, compute ony latency = %8.7f ms, per element = %8.7f microseconds.\n", n,
      time_iter, 1000.0*time_iter / (double)n); 
  NEW_CUDA_CHECK(hipMemcpy(localInstances, gpuInstances, sizeof(tuple_mfq_ti) * n, hipMemcpyDeviceToHost));
  
  for (int i = 0; i < 1; i++) {
    x0.emplace_back(to_fq(x[i]));
    x1.emplace_back(to_fq(y[i]));
    std::memcpy(z, x[i], num_bytes * sizeof(uint8_t));
    int carry = mpn_sub(z, y[i], num_bytes);
    fprintf(debug_file, "\n REF SUB, carry = %d:\n", carry);
    fprint_uint8_array(debug_file, (uint8_t*)z, num_bytes); 
    fprintf(debug_file, "\n MY SUB, carry = %d:\n", localInstances[i].carry);
    fprint_uint8_array(debug_file, (uint8_t*)localInstances[i].x, num_bytes); 
    if (check((uint8_t*) z, (uint8_t*)localInstances[i].x, io_bytes_per_elem)) {
      printf("\n TEST FAILED.");
      fprintf(debug_file, "\n TEST FAILED.");
    }
  }

  // free memory
  hipFree(gpuInstances);
  free(localInstances);
  free(z);
}

// We test basic big int addition by a0 + a1 for a fq2 element.
void test_fq_sub(std::vector<uint8_t*> x, std::vector<uint8_t*> y, int num_bytes, FILE* debug_file) {
  mnt4753_pp::init_public_params();
  mnt6753_pp::init_public_params();

  std::vector<Fq<mnt4753_pp>> x0;
  std::vector<Fq<mnt4753_pp>> x1;
  cgbn_error_report_t *report;
  NEW_CUDA_CHECK(cgbn_error_report_alloc(&report));

  int tpb = TPB;
  // printf("\n Threads per block =%d", tpb);
  int IPB = TPB/TPI;

  int n = x.size();
  tuple_mfq_ti* gpuInstances;
  tuple_mfq_ti* localInstances;
  fprintf(debug_file, "\n size of fq2_t:%d", sizeof(tuple_mfq_ti));
  localInstances = (tuple_mfq_ti*) calloc(n, sizeof(tuple_mfq_ti));
  NEW_CUDA_CHECK(hipSetDevice(0));
  NEW_CUDA_CHECK(hipMalloc((void **)&gpuInstances, sizeof(tuple_mfq_ti)*n));
  load_mnt4_modulus();
  
  for (int i = 0; i < n; i++) {
      std::memcpy((void*)localInstances[i].x, (void*)x[i], num_bytes);
      std::memcpy((void*)localInstances[i].y, (void*)y[i], num_bytes);
  }
  
  NEW_CUDA_CHECK(hipMemcpy(gpuInstances, localInstances, sizeof(tuple_mfq_ti) * n, hipMemcpyHostToDevice));
  //for (int i = 0; i < n; i++) {
  //    NEW_CUDA_CHECK(hipMemcpy(gpuInstances[i].a0, x[i], num_bytes, hipMemcpyHostToDevice));
  //    NEW_CUDA_CHECK(hipMemcpy(gpuInstances[i].a1, y[i], num_bytes, hipMemcpyHostToDevice));
  //}

  uint32_t num_blocks = (n + IPB-1)/IPB;
  clock_t start, end;
  double time_iter = 0.0;

  start = clock();
  fq_sub_kernel<<<num_blocks, TPB>>>(gpuInstances, n, mnt4_modulus_device);
  NEW_CUDA_CHECK(hipDeviceSynchronize());
  end = clock();
  time_iter = ((double) end-start) * 1000.0 / CLOCKS_PER_SEC;
  fprintf(debug_file, "\n num_elements = %d, compute ony latency = %8.7f ms, per element = %8.7f microseconds.\n", n,
      time_iter, 1000.0*time_iter / (double)n); 
  printf("\n num_elements = %d, compute ony latency = %8.7f ms, per element = %8.7f microseconds.\n", n,
      time_iter, 1000.0*time_iter / (double)n); 
  NEW_CUDA_CHECK(hipMemcpy(localInstances, gpuInstances, sizeof(tuple_mfq_ti) * n, hipMemcpyDeviceToHost));
  
  for (int i = 0; i < n; i++) {
    x0.emplace_back(to_fq(x[i]));
    x1.emplace_back(to_fq(y[i]));
    Fq<mnt4753_pp> out = x0[i] - x1[i];
    fprintf(debug_file, "\n REF ADD:\n");
    fprint_fq(debug_file, out); 
    fprintf(debug_file, "\n MY ADD:\n");
    fprint_uint8_array(debug_file, (uint8_t*)localInstances[i].x, num_bytes); 
    if (check((uint8_t*) out.mont_repr.data, (uint8_t*)localInstances[i].x, io_bytes_per_elem)) {
      printf("\n TEST FAILED.");
      fprintf(debug_file, "\n TEST FAILED.");
    }
  }

  // free memory
  hipFree(gpuInstances);
  free(localInstances);
}

// We test basic big int addition by a0 + a1 for a fq2 element.
void test_fq_mont_mul(std::vector<uint8_t*> x, std::vector<uint8_t*> y, int num_bytes, FILE* debug_file) {
  mnt4753_pp::init_public_params();
  mnt6753_pp::init_public_params();

  std::vector<Fq<mnt4753_pp>> x0;
  std::vector<Fq<mnt4753_pp>> x1;
  cgbn_error_report_t *report;
  NEW_CUDA_CHECK(cgbn_error_report_alloc(&report));

  int tpb = TPB;
  // printf("\n Threads per block =%d", tpb);
  int IPB = TPB/TPI;

  int n = x.size();
  triple_mfq_ti* gpuInstances;
  triple_mfq_ti* localInstances;
  fprintf(debug_file, "\n size of fq2_t:%d", sizeof(triple_mfq_ti));
  localInstances = (triple_mfq_ti*) calloc(n, sizeof(triple_mfq_ti));
  NEW_CUDA_CHECK(hipSetDevice(0));
  NEW_CUDA_CHECK(hipMalloc((void **)&gpuInstances, sizeof(triple_mfq_ti)*n));
  load_mnt4_modulus();
  
  for (int i = 0; i < n; i++) {
      std::memcpy((void*)localInstances[i].x, (void*)x[i], num_bytes);
      std::memcpy((void*)localInstances[i].y, (void*)y[i], num_bytes);
      std::memset((void*)localInstances[i].r, 0, num_bytes);
  }
  
  NEW_CUDA_CHECK(hipMemcpy(gpuInstances, localInstances, sizeof(triple_mfq_ti) * n, hipMemcpyHostToDevice));
  //for (int i = 0; i < n; i++) {
  //    NEW_CUDA_CHECK(hipMemcpy(gpuInstances[i].a0, x[i], num_bytes, hipMemcpyHostToDevice));
  //    NEW_CUDA_CHECK(hipMemcpy(gpuInstances[i].a1, y[i], num_bytes, hipMemcpyHostToDevice));
  //}

  uint32_t num_blocks = (n + IPB-1)/IPB;
  clock_t start, end;
  double time_iter = 0.0;

  start = clock();
  fq_mont_mul_kernel<<<num_blocks, TPB>>>(gpuInstances, n, mnt4_modulus_device);
  NEW_CUDA_CHECK(hipDeviceSynchronize());
  end = clock();
  time_iter = ((double) end-start) * 1000.0 / CLOCKS_PER_SEC;
  fprintf(debug_file, "\n num_elements = %d, compute ony latency = %8.7f ms, per element = %8.7f microseconds.\n", n,
      time_iter, 1000.0*time_iter / (double)n); 
  printf("\n num_elements = %d, compute ony latency = %8.7f ms, per element = %8.7f microseconds.\n", n,
      time_iter, 1000.0*time_iter / (double)n); 
  NEW_CUDA_CHECK(hipMemcpy(localInstances, gpuInstances, sizeof(triple_mfq_ti) * n, hipMemcpyDeviceToHost));
  
  for (int i = 0; i < n; i++) {
    x0.emplace_back(to_fq(x[i]));
    x1.emplace_back(to_fq(y[i]));
    Fq<mnt4753_pp> out = x0[i] * x1[i];
    fprintf(debug_file, "\n X_org:\n");
    fprint_fq(debug_file, x0[i]); 
    fprintf(debug_file, "\n X:\n");
    fprint_uint8_array(debug_file, (uint8_t*)localInstances[i].x, num_bytes); 
    fprintf(debug_file, "\n Y:\n");
    fprint_uint8_array(debug_file, (uint8_t*)localInstances[i].y, num_bytes); 
    fprintf(debug_file, "\n Y_orig:\n");
    fprint_uint8_array(debug_file, (uint8_t*)y[i], num_bytes); 
    fprintf(debug_file, "\n REF ADD:\n");
    fprint_fq(debug_file, out); 
    fprintf(debug_file, "\n MY ADD:\n");
    fprint_uint8_array(debug_file, (uint8_t*)localInstances[i].r, num_bytes); 
    if (check((uint8_t*) out.mont_repr.data, (uint8_t*)localInstances[i].r, io_bytes_per_elem)) {
      printf("\n TEST FAILED.");
      fprintf(debug_file, "\n TEST FAILED.");
    }
  }

  // free memory
  hipFree(gpuInstances);
  free(localInstances);
}

// We test basic big int addition by a0 + a1 for a fq2 element.
void test_fq_add(std::vector<uint8_t*> x, std::vector<uint8_t*> y, int num_bytes, FILE* debug_file) {
  mnt4753_pp::init_public_params();
  mnt6753_pp::init_public_params();

  std::vector<Fq<mnt4753_pp>> x0;
  std::vector<Fq<mnt4753_pp>> x1;
  cgbn_error_report_t *report;
  NEW_CUDA_CHECK(cgbn_error_report_alloc(&report));

  int tpb = TPB;
  // printf("\n Threads per block =%d", tpb);
  int IPB = TPB/TPI;

  int n = x.size();
  tuple_mfq_ti* gpuInstances;
  tuple_mfq_ti* localInstances;
  fprintf(debug_file, "\n size of fq2_t:%d", sizeof(tuple_mfq_ti));
  localInstances = (tuple_mfq_ti*) calloc(n, sizeof(tuple_mfq_ti));
  NEW_CUDA_CHECK(hipSetDevice(0));
  NEW_CUDA_CHECK(hipMalloc((void **)&gpuInstances, sizeof(tuple_mfq_ti)*n));
  load_mnt4_modulus();
  
  for (int i = 0; i < n; i++) {
      std::memcpy((void*)localInstances[i].x, (void*)x[i], num_bytes);
      std::memcpy((void*)localInstances[i].y, (void*)y[i], num_bytes);
  }
  
  NEW_CUDA_CHECK(hipMemcpy(gpuInstances, localInstances, sizeof(tuple_mfq_ti) * n, hipMemcpyHostToDevice));
  //for (int i = 0; i < n; i++) {
  //    NEW_CUDA_CHECK(hipMemcpy(gpuInstances[i].a0, x[i], num_bytes, hipMemcpyHostToDevice));
  //    NEW_CUDA_CHECK(hipMemcpy(gpuInstances[i].a1, y[i], num_bytes, hipMemcpyHostToDevice));
  //}

  uint32_t num_blocks = (n + IPB-1)/IPB;
  clock_t start, end;
  double time_iter = 0.0;

  start = clock();
  fq_add_kernel<<<num_blocks, TPB>>>(gpuInstances, n, mnt4_modulus_device);
  NEW_CUDA_CHECK(hipDeviceSynchronize());
  end = clock();
  time_iter = ((double) end-start) * 1000.0 / CLOCKS_PER_SEC;
  fprintf(debug_file, "\n num_elements = %d, compute ony latency = %8.7f ms, per element = %8.7f microseconds.\n", n,
      time_iter, 1000.0*time_iter / (double)n); 
  printf("\n num_elements = %d, compute ony latency = %8.7f ms, per element = %8.7f microseconds.\n", n,
      time_iter, 1000.0*time_iter / (double)n); 
  NEW_CUDA_CHECK(hipMemcpy(localInstances, gpuInstances, sizeof(tuple_mfq_ti) * n, hipMemcpyDeviceToHost));
  
  for (int i = 0; i < n; i++) {
    x0.emplace_back(to_fq(x[i]));
    x1.emplace_back(to_fq(y[i]));
    Fq<mnt4753_pp> out = x0[i] + x1[i];
    fprintf(debug_file, "\n REF ADD:\n");
    fprint_fq(debug_file, out); 
    fprintf(debug_file, "\n MY ADD:\n");
    fprint_uint8_array(debug_file, (uint8_t*)localInstances[i].x, num_bytes); 
    if (check((uint8_t*) out.mont_repr.data, (uint8_t*)localInstances[i].x, io_bytes_per_elem)) {
      printf("\n TEST FAILED.");
      fprintf(debug_file, "\n TEST FAILED.");
    }
  }

  // free memory
  hipFree(gpuInstances);
  free(localInstances);
}

int main(int argc, char* argv[]) {
  printf("\nMain program. argc = %d \n", argc);

  //loadrun_fq_add(input_a, "debug_log");
  //loadrun_fq_op<add_fq_op>(input_a, "debug_log");
  //loadrun_fq_op<sub_fq_op>(input_a, "debug_log");
  loadrun_fq_op<mul_by13_fq_op>(input_a, "debug_log");
  loadrun_fq_op<mont_mul_fq_op>(input_a, "debug_log");

  //loadrun_fq_op<sub_newtest>(input_a, "debug_log");
}

